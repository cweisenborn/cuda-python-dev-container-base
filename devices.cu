#include <stdio.h> 

int main() {
  int devices;
  hipGetDeviceCount(&devices);
  
  for (int i = 0; i < devices; i++) {
    hipDeviceProp_t deviceInfo;
    hipGetDeviceProperties(&deviceInfo, i);
    
    printf("Device Number: %d\n", i);
    printf("- Device Name: %s\n", deviceInfo.name);
    printf("- Memory Clock Rate (KHz): %d\n", deviceInfo.memoryClockRate);
    printf("- Memory Bus Width (bits): %d\n", deviceInfo.memoryBusWidth);
  }
}